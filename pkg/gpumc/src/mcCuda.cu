#include "hip/hip_runtime.h"
#include <math.h>
#include <R.h>

#define TPB 1024

__device__ double mcCalc(double x, double *d_samps, int S)
{
	double total = 0.0f;
	for (int i = 0; i < S; i++)
	{
		total += cos(x * d_samps[i]);
	}
	return total / S;
}

__global__ void mcKernel(double *d_vec, double *d_samps, double *d_mat, int N, int S)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N) return;
	const double x = d_vec[i];
	d_mat[i] = mcCalc(x, d_samps, S);
}


// Helper function for using CUDA to add vectors in parallel.
extern "C" void mcCuda(double *vec, double *samps, double *mat, int N, int S)
{

    double *d_vec = 0;
    double *d_samps = 0;
    double *d_mat = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&d_vec, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        error("hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_samps, S * sizeof(double));
    if (cudaStatus != hipSuccess) {
        error("hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_mat, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        error("hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_vec, vec, N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        error("hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_samps, samps, S * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        error("hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with TPB threads per block.
    mcKernel<<<(N+TPB-1)/TPB, TPB>>>(d_vec, d_samps, d_mat, N, S);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        error("mcKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        error("hipDeviceSynchronize returned error code %d after launching mcKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(mat, d_mat, N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        error("hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(d_vec);
    hipFree(d_samps);
    hipFree(d_mat);

    hipDeviceReset();

}
